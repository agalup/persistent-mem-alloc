#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <cassert>
#include <algorithm>
#include <thread>
#include <chrono>
#include <any>

#include "PerformanceMeasure.cuh"
#include "hip/hip_runtime.h"
#include "pmm-utils.cuh"

using namespace std;

extern "C" {
/*
__device__
void _request_processing(
        int request_id, 
        volatile int* request_signal,
        volatile int* request_counter,
        volatile int* request_ids, 
        volatile int* lock){

    // SEMAPHORE
    acquire_semaphore((int*)(lock + request_id));
    if (request_signal[request_id] == MOCK){
        if (request_ids[request_id] == -1){
            request_ids[request_id] = atomicAdd((int*)&request_counter[0], 1);
        }
        atomicExch((int*)&request_signal[request_id], DONE);
        __threadfence();
    }
    release_semaphore((int*)(lock + request_id));
    // SEMAPHORE
}*/

//producer
__global__
void mem_manager(volatile int* exit_signal, 
        volatile int* mm_started,
        volatile int* requests_number, 
        volatile int* request_counter,
        volatile int* request_signal, 
        volatile int* request_ids, 
        volatile int* lock){

    mm_started[0] = 1;
    uint32_t ret;
    asm("mov.u32 %0, %smid;" : "=r"(ret));
    auto es = exit_signal[0];
    if (blockIdx.x > 30)
        printf("1: memory manager: block id %d/%d, smid %d, exit_signal = %d\n", blockIdx.x, gridDim.x, ret, es);

    while (! exit_signal[0] ){
        asm("mov.u32 %0, %smid;" : "=r"(ret));
        if (blockIdx.x > 30)
            printf("2: memory manager: block id %d/%d, smid %d\n", blockIdx.x, gridDim.x, ret);
        for (int request_id = thid(); !exit_signal[0] && 
                request_id < requests_number[0]; 
                request_id += blockDim.x*gridDim.x){
                uint32_t ret;
                asm("mov.u32 %0, %smid;" : "=r"(ret));
                if (blockIdx.x > 30)
                    printf("3: memory manager: request id %d/%d, smid %d\n", request_id, requests_number[0], ret);
        }
        __threadfence();
        break;
    }
}
/*
__device__
void post_request(request_type type,
                  volatile int* lock,
                  volatile int* request_signal){

    debug("request %d, block %d\n", thid(), blockIdx.x);
    
    __threadfence();
    // SEMAPHORE
    //acquire_semaphore((int*)lock, thid());
    //int* ptr = (int*)(lock + thid());
    acquire_semaphore((int*)(lock + thid()));
    // SIGNAL update
    atomicExch((int*)&request_signal[thid()], type);
    //release_semaphore((int*)lock, thid());
    release_semaphore((int*)(lock + thid()));

    __threadfence();
    // SEMAPHORE
}*/
/*
__device__
void request_processed(request_type type,
                      volatile int* lock,
                      volatile int* request_signal){
    //int thid = blockDim.x * blockIdx.x + threadIdx.x;
    // SEMAPHORE
    __threadfence();
    //acquire_semaphore((int*)lock, thid());
    //int* ptr = (int*)(lock + thid());
    acquire_semaphore((int*)(lock + thid()));
    switch (type){
        case MOCK:
            //req_id = request_id[thid()];
            break;
        default:
            //printf("error\n");
            break;
    }
    // SIGNAL update
    request_signal[thid()] = request_empty;
    //release_semaphore((int*)lock, thid());
    release_semaphore((int*)(lock + thid()));
    //debug("request %d, block %d done\n", thid(), blockIdx.x);
    __threadfence();
    // SEMAPHORE
}*/

__device__
void request(request_type type,
        volatile int* exit_signal,
        volatile int* request_signal,
        volatile int* lock
        ){

    // POST REQUEST: wait for success
    while (!exit_signal[0]){
        uint32_t ret;
        if (thid() > 30){
            asm("mov.u32 %0, %smid;" : "=r"(ret));
            printf("requested by %d/%d, smid %d\n", thid(), blockDim.x*gridDim.x, ret);
        }
        break;
        /*if (request_signal[thid()] == request_empty){
            //post_request(type, lock, request_signal);
        }
        __threadfence();*/
    }

    // REQUEST PROCESSED
    // int it = 0;
    // wait for success
    while (!exit_signal[0]){
        break;
        /*
        if (request_signal[thid()] == request_done){
            //request_processed(type, lock, request_signal);
            break;
        }
        __threadfence();*/
    }
}

}
