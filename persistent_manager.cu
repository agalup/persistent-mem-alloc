#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <cassert>
#include <algorithm>
#include <thread>
#include <chrono>
#include <any>

#include "PerformanceMeasure.cuh"
#include "hip/hip_runtime.h"
#include "pmm-utils.cuh"

using namespace std;

extern "C" {

__device__
void _request_processing(
        int request_id, 
        //volatile int* exit_signal,
        volatile int* request_signal,
        volatile int* request_counter,
        volatile int* request_ids, 
        //volatile int** request_dest, 
        //volatile int** d_memory,
        //volatile int* request_mem_size,
        volatile int* lock){

    // SEMAPHORE
    //acquire_semaphore((int*)lock, request_id);
    //int* ptr = (int*)(lock + request_id);
    //acquire_semaphore(ptr);
    acquire_semaphore((int*)(lock + request_id));
    //debug("mm: request recieved %d\n", request_id); 
    //auto addr_id = request_ids[request_id];

    if (request_signal[request_id] == MOCK){
        request_ids[request_id] = (request_ids[request_id] == -1 ? atomicAdd((int*)&request_counter[0], 1) :
        request_ids[request_id]);
        atomicExch((int*)&request_signal[request_id], DONE);
        __threadfence();
    }
    
    /*switch (request_signal[request_id]){

        case MOCK:
            //if (addr_id == -1){
                //addr_id = atomicAdd((int*)&request_counter[0], 1);
                //request_ids[request_id] = addr_id;
            if (request_ids[request_id] == -1){
                request_ids[request_id] = atomicAdd((int*)&request_counter[0], 1);
    //        }else{
      //          printf("error\n");
            }
            atomicExch((int*)&request_signal[request_id], request_done);
            __threadfence();
            break;

        default:
            break;
     //       printf("request processing fail\n");

    }*/

    //release_semaphore((int*)lock, request_id);
    release_semaphore((int*)(lock + request_id));
    // SEMAPHORE
}



//producer
__global__
void mem_manager(volatile int* exit_signal, 
        volatile int* mm_started,
        volatile int* requests_number, 
        volatile int* request_counter,
        volatile int* request_signal, 
        volatile int* request_ids, 
        volatile int** request_dest,
        volatile int** d_memory,
        volatile int* request_mem_size,
        volatile int* lock){
    //int thid = blockIdx.x * blockDim.x + threadIdx.x;

    mm_started[0] = 1;
    
    while (! exit_signal[0] ){
        for (int request_id = thid(); !exit_signal[0] && request_id < requests_number[0]; 
                request_id += blockDim.x*gridDim.x){

            __threadfence();
            if (request_signal[request_id] == MOCK){
                _request_processing(request_id, /*exit_signal,*/ request_signal, 
                                    request_counter, request_ids,/* request_dest,*/
                                    /*d_memory, request_mem_size,*/ lock);
                __threadfence();
                debug("mm: request done %d\n", request_id);
            }
        }
        __threadfence();
    }
}

__device__
void post_request(request_type type,
                  //volatile int** dest,
                  volatile int* lock,
                  //volatile int* request_mem_size,
                  //volatile int* request_id,
                  volatile int* request_signal
                  //,volatile int** request_dest,
                  //volatile int* exit_signal,
                  //int size_to_alloc
                  ){

    //int thid = blockDim.x * blockIdx.x + threadIdx.x;
    debug("request %d, block %d\n", thid(), blockIdx.x);
    
    __threadfence();
    // SEMAPHORE
    //acquire_semaphore((int*)lock, thid());
    //int* ptr = (int*)(lock + thid());
    acquire_semaphore((int*)(lock + thid()));
    // SIGNAL update
    atomicExch((int*)&request_signal[thid()], type);
    //release_semaphore((int*)lock, thid());
    release_semaphore((int*)(lock + thid()));

    __threadfence();
    // SEMAPHORE
}

__device__
void request_processed(request_type type,
                      volatile int* lock,
                      /*volatile int* request_id,
                      volatile int* exit_signal,
                      volatile int** d_memory,
                      volatile int** dest,*/
                      volatile int* request_signal
                      /*,volatile int** request_dest*/){
                      //int& req_id){
    //int thid = blockDim.x * blockIdx.x + threadIdx.x;
    // SEMAPHORE
    __threadfence();
    //acquire_semaphore((int*)lock, thid());
    //int* ptr = (int*)(lock + thid());
    acquire_semaphore((int*)(lock + thid()));
    switch (type){
        case MOCK:
            //req_id = request_id[thid()];
            break;
        default:
            //printf("error\n");
            break;
    }
    // SIGNAL update
    request_signal[thid()] = request_empty;
    //release_semaphore((int*)lock, thid());
    release_semaphore((int*)(lock + thid()));
    //debug("request %d, block %d done\n", thid(), blockIdx.x);
    __threadfence();
    // SEMAPHORE
}

__device__
void request(request_type type,
        volatile int* exit_signal,
        //volatile int** d_memory,
        //volatile int** dest,
        volatile int* request_signal,
        //volatile int* request_mem_size, 
        //volatile int* request_id,
        //volatile int** request_dest,
        volatile int* lock
        //, int size_to_alloc
        ){

    // POST REQUEST: wait for success
    while (!exit_signal[0]){
        if (request_signal[thid()] == request_empty){
            post_request(type, /*dest,*/ lock, /*request_mem_size, 
                        request_id, */ request_signal /*,request_dest, 
                        exit_signal, size_to_alloc*/);
            break;
        }
        __threadfence();
    }
    ///__threadfence();

    // REQUEST PROCESSED
    // int it = 0;
    // wait for success
    while (!exit_signal[0]){
        if (request_signal[thid()] == request_done){
            request_processed(type, lock,/* request_id, exit_signal, d_memory, 
                        dest,*/ request_signal/*, request_dest*/);//, -1);
            break;
        }
        __threadfence();
    }
}


/*
void check_persistent_kernel_results(int* exit_signal, 
                   int* exit_counter, 
                   int block_size, 
                   int app_grid_size, 
                   RequestType& requests, 
                   int requests_num,
                   bool& finish){

    // Check results
    int old_counter = -1;
    long long int iter = 0;
    long long int time_limit = 1000000000;
    //printf("waiting till allocations are done\n");
    while (iter < time_limit){
        std::this_thread::sleep_for(std::chrono::microseconds(1));
        //if (iter%60 == 0)
        //    printf("%lld min, exit counter %d\n", iter/60, exit_counter[0]);
        // Check if all allocations are done
        if (exit_counter[0] == block_size*app_grid_size){
            GUARD_CU(hipStreamSynchronize(0));
            GUARD_CU(hipPeekAtLastError());
            finish = true;
            break;
        }else{
            GUARD_CU(hipPeekAtLastError());
            if (exit_counter[0] != old_counter){
                old_counter = exit_counter[0];
                //printf("%d\n", old_counter);
                iter = 0;
            }
            ++iter;
        }
        if (iter >= time_limit){
            // Start mm and app again
            printf("time limit exceed, break\n");
            fflush(stdout);
            *exit_signal = 1;
            GUARD_CU(hipDeviceSynchronize());
            GUARD_CU(hipPeekAtLastError());
        }
    }
    GUARD_CU(hipPeekAtLastError());
}*/

}
