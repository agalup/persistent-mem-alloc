#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <cassert>
#include <algorithm>
#include <thread>
#include <chrono>
#include <any>

#include "device/Ouroboros_impl.cuh"
#include "device/MemoryInitialization.cuh"
#include "InstanceDefinitions.cuh"
#include "PerformanceMeasure.cuh"
#include "Utility.h"
#include "hip/hip_runtime.h"
#include "pmm-utils.cuh"

//#include "src/gpu_hash_table.cuh"

using namespace std;

extern "C" {

__global__
    void mem_free(volatile int** d_memory, 
            volatile int* request_id, 
            MemoryManagerType* mm, 
            volatile int* requests_num){
    int thid = blockDim.x * blockIdx.x + threadIdx.x;
    if (thid >= requests_num[0]){
        return;
    }
    __threadfence();
    if (d_memory[thid]){
        printf("sync error: %d was not released before\n", thid);
        mm->free((void*)d_memory[thid]);
    }
}

__device__
void _request_processing(
        int request_id, 
        volatile int* exit_signal,
        RequestType requests,
       /* volatile int* request_signal,
        volatile int* request_counter,
        volatile int* request_ids, 
        volatile int** request_dest, */
        MemoryManagerType* mm
       /*, volatile int** d_memory,
        volatile int* request_mem_size,
        volatile int* lock*/){
    debug("request processing!\n");

    // SEMAPHORE
    acquire_semaphore((int*)requests.lock, request_id);
    debug("MEM MANAGER %s: thid %d, block ID %d, warp ID %d, lane ID %d, sm ID %d\n", __FUNCTION__, request_id, blockIdx.x, warp_id(), lane_id(), sm_id());
    //debug("mm: request recieved %d\n", request_id); 
    auto addr_id = requests.request_id[request_id];
    int request_status;
    
    switch (requests.request_signal[request_id]){

        case MALLOC:
            if (addr_id == -1){
                addr_id = atomicAdd((int*)&requests.request_counter[0], 1);
                requests.request_id[request_id] = addr_id;
            }else{
                assert(requests.d_memory[addr_id] == NULL);
            }
            __threadfence();
            requests.d_memory[addr_id] = reinterpret_cast<volatile int*>
                (mm->malloc(4+requests.request_mem_size[request_id]));
            __threadfence();
            assert(requests.d_memory[addr_id]);
            requests.d_memory[addr_id][0] = 0;
            requests.request_dest[request_id] = &requests.d_memory[addr_id][1];
            atomicExch((int*)&requests.request_signal[request_id], request_done);
            break;

        case FREE:
            assert(requests.d_memory[addr_id]);
            if (requests.d_memory[addr_id][0] != 0)
                printf("d_memory{%d} = %d\n", addr_id, requests.d_memory[addr_id][0]);
            assert(requests.d_memory[addr_id][0] == 0);
            request_status = requests.d_memory[addr_id][0] - 1;
            requests.d_memory[addr_id][0] -= 1;
            requests.request_dest[request_id] = NULL;
            assert(requests.d_memory[addr_id][0] == -1);
            if (request_status < 0){
                atomicExch((int*)&requests.request_signal[request_id], request_gc);
            }else{
                assert(1);
                printf("should not be here!\n");
                atomicExch((int*)&requests.request_signal[request_id], request_done);
            }
            break;

        case GC:
            assert(requests.d_memory[addr_id]);
            assert(requests.d_memory[addr_id][0] == -1);
            __threadfence();
            mm->free((void*)requests.d_memory[addr_id]);
            __threadfence();
            requests.d_memory[addr_id] = NULL;
            atomicExch((int*)&requests.request_signal[request_id], request_done);
            break;

        default:
            printf("request processing fail\n");

    }
    release_semaphore((int*)requests.lock, request_id);
    // SEMAPHORE
}

/*__global__
void garbage_collector(
                       volatile int* exit_signal,
                       volatile int* gc_started,
                       volatile int* requests_number, 
                       volatile int* request_counter,
                       volatile int* request_signal, 
                       volatile int* request_ids, 
                       volatile int* request_mem_size,
                       volatile int** request_dest,
                       volatile int** d_memory,
                       volatile int* lock,
                       MemoryManagerType* mm
                       ){
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    gc_started[0] = 1;
    while (! exit_signal[0]){
        debug("hello gc! %d\n", thid);
        assert(requests_number);
        assert(request_signal);
        for (int request_id = thid; !exit_signal[0] && 
                request_id < requests_number[0]; 
                request_id += blockDim.x*gridDim.x){
            __threadfence();
            if ((request_signal[request_id]) == GC){
                _request_processing(request_id, exit_signal, 
                                    request_signal,
                                    request_counter, 
                                    request_ids, 
                                    request_dest,
                                    mm, d_memory, 
                                    request_mem_size, 
                                    lock);
                __threadfence();
            }
        }
        __threadfence();
    }
}*/

__global__
void garbage_collector(
                       volatile int* exit_signal,
                       volatile int* gc_started,
                       RequestType requests,
                       MemoryManagerType* mm
                       ){
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    gc_started[0] = 1;
    while (! exit_signal[0]){
        debug("hello gc! %d\n", thid);
        //assert(requests_number);
        //assert(request_signal);
        for (int request_id = thid; !exit_signal[0] && 
                request_id < requests.requests_number[0]; 
                request_id += blockDim.x*gridDim.x){
            __threadfence();
            if ((requests.request_signal[request_id]) == GC){
                _request_processing(request_id, exit_signal, 
                                    requests, mm);
                __threadfence();
            }
        }
        __threadfence();
    }
}



//producer
__global__
void mem_manager(volatile int* exit_signal, 
                volatile int* mm_started,
                RequestType requests,
                /*volatile int* requests_number, 
                volatile int* request_counter,
                volatile int* request_signal, 
                volatile int* request_ids, 
                volatile int* request_mem_size,
                volatile int** request_dest,
                volatile int** d_memory,
                volatile int* lock,*/
                MemoryManagerType* mm
                ){
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    mm_started[0] = 1;
    while (! exit_signal[0]){
        debug("hello mm %d, request no %d!\n", thid, requests_number[0]);
        for (int request_id = thid; !exit_signal[0] && 
                request_id < requests.requests_number[0]; 
                request_id += blockDim.x*gridDim.x){

            __threadfence();
            if ((requests.request_signal[request_id]) == MALLOC or 
                (requests.request_signal[request_id]) == FREE){
                _request_processing(request_id, exit_signal, 
                                    requests, mm);
                /*
                                    requests.request_signal, 
                                    requests.request_counter, 
                                    requests.request_id, 
                                    requests.request_dest,
                                    mm, requests.d_memory, 
                                    requests.request_mem_size, 
                                    requests.lock);*/
                __threadfence();
                debug("mm: request done %d\n", request_id);
            }
        }
        /*
        for (int request_id = thid; !exit_signal[0] && 
                request_id < requests_number[0]; 
                request_id += blockDim.x*gridDim.x){

            __threadfence();
            if ((request_signal[request_id]) == MALLOC or 
                (request_signal[request_id]) == FREE){
                _request_processing(request_id, exit_signal, 
                                    request_signal, 
                                    request_counter, 
                                    request_ids, 
                                    request_dest,
                                    mm, d_memory, 
                                    request_mem_size, 
                                    lock);
                __threadfence();
                debug("mm: request done %d\n", request_id);
            }
        }*/
        __threadfence();
    }
}

/*__device__
void post_request(request_type type,
                  volatile int** dest,
                  volatile int* lock,
                  volatile int* request_mem_size,
                  volatile int* request_id,
                  volatile int* request_signal,
                  volatile int** request_dest,
                  volatile int* exit_signal,
                  int size_to_alloc){

    int thid = blockDim.x * blockIdx.x + threadIdx.x;
    
    __threadfence();
    // SEMAPHORE
    acquire_semaphore((int*)lock, thid);
    if (type == MALLOC){
        request_mem_size[thid] = size_to_alloc;
    }
    // SIGNAL update
    atomicExch((int*)&request_signal[thid], type);
    debug("APP %s: thid %d, block ID %d, warp ID %d, lane ID %d, sm ID %d\n", __FUNCTION__, thid, blockIdx.x, warp_id(), lane_id(), sm_id());
    release_semaphore((int*)lock, thid);
    __threadfence();
    // SEMAPHORE
}

__device__
void request_processed(request_type type,
                      volatile int* lock,
                      volatile int* request_id,
                      volatile int* exit_signal,
                      volatile int** d_memory,
                      volatile int** dest,
                      volatile int* request_signal,
                      volatile int** request_dest,
                      int& req_id){
    int thid = blockDim.x * blockIdx.x + threadIdx.x;
    // SEMAPHORE
    __threadfence();
    acquire_semaphore((int*)lock, thid);
    debug("APP %s: thid %d, block ID %d, warp ID %d, lane ID %d, sm ID %d\n", __FUNCTION__, thid, blockIdx.x, warp_id(), lane_id(), sm_id());
    switch (type){
        case MALLOC:
            req_id = request_id[thid];
            if (req_id >= 0 && !exit_signal[0]) {
                *dest = request_dest[thid];
                assert(d_memory[req_id] != NULL);
                if (d_memory[req_id][0] != 0)
                    printf("d_memory[%d] = %d\n", req_id, d_memory[req_id][0]);
                //assert(d_memory[req_id][0] == 0);
                assert(*dest != NULL);
                assert(request_dest[thid] == *dest);
            }
            break;
        case FREE:
            //assert(d_memory[req_id] == NULL);
            break;
        case GC:
            //assert(d_memory[req_id] == NULL);
            break;
        default:
            printf("error\n");
            break;
    }
    request_signal[thid] = request_empty;
    release_semaphore((int*)lock, thid);
    __threadfence();
    // SEMAPHORE
}
*/
__device__
void post_request(request_type type,
                  volatile int* exit_signal,
                  RequestType& requests,
                  volatile int** dest,
                  int size_to_alloc){

    int thid = blockDim.x * blockIdx.x + threadIdx.x;
    
    __threadfence();
    // SEMAPHORE
    acquire_semaphore((int*)requests.lock, thid);
    if (type == MALLOC){
        requests.request_mem_size[thid] = size_to_alloc;
    }
    // SIGNAL update
    atomicExch((int*)&requests.request_signal[thid], type);
    debug("APP %s: thid %d, block ID %d, warp ID %d, lane ID %d, sm ID %d\n", __FUNCTION__, thid, blockIdx.x, warp_id(), lane_id(), sm_id());
    release_semaphore((int*)requests.lock, thid);
    __threadfence();
    // SEMAPHORE
}

__device__
void request_processed(request_type type,
                      volatile int* exit_signal,
                      RequestType& requests,
                      volatile int** dest,
                      int& req_id){
    int thid = blockDim.x * blockIdx.x + threadIdx.x;
    // SEMAPHORE
    __threadfence();
    acquire_semaphore((int*)requests.lock, thid);
    debug("APP %s: thid %d, block ID %d, warp ID %d, lane ID %d, sm ID %d\n", __FUNCTION__, thid, blockIdx.x, warp_id(), lane_id(), sm_id());
    switch (type){
        case MALLOC:
            req_id = requests.request_id[thid];
            if (req_id >= 0 && !exit_signal[0]) {
                *dest = requests.request_dest[thid];
                assert(requests.d_memory[req_id] != NULL);
                if (requests.d_memory[req_id][0] != 0)
                    printf("d_memory[%d] = %d\n", req_id, requests.d_memory[req_id][0]);
                //assert(d_memory[req_id][0] == 0);
                assert(*dest != NULL);
                assert(requests.request_dest[thid] == *dest);
            }
            break;
        case FREE:
            //assert(d_memory[req_id] == NULL);
            break;
        case GC:
            //assert(d_memory[req_id] == NULL);
            break;
        default:
            printf("error\n");
            break;
    }
    requests.request_signal[thid] = request_empty;
    release_semaphore((int*)requests.lock, thid);
    __threadfence();
    // SEMAPHORE
}
/*
__device__
void request(request_type type,
        volatile int* exit_signal,
        volatile int** d_memory,
        volatile int** dest,
        volatile int* request_signal,
        volatile int* request_mem_size, 
        volatile int* request_id,
        volatile int** request_dest,
        volatile int* lock,
        int size_to_alloc){

    int thid = blockDim.x * blockIdx.x + threadIdx.x;
    int req_id = -1;
    // wait for success
    while (!exit_signal[0]){
        if (request_signal[thid] == request_empty){
            post_request(type, dest, lock, request_mem_size, request_id, 
                        request_signal, request_dest, exit_signal, size_to_alloc);
            break;
        }
        __threadfence();
    }

    __threadfence();

    //int it = 0;
    // wait for success
    while (!exit_signal[0]){
        //char* type_ = new char[10];
        //char* state = new char[10];
        //if (++it > 1000){
        //    if (type == MALLOC) type_ = "MALLOC"; else type_ = "FREE";
        //    switch (request_signal[thid]){
        //        case request_empty:  state = "EMPTY";  break;
        //        case request_done:   state = "DONE";   break;
        //        case request_malloc: state = "MALLOC"; break;
        //        case request_free:   state = "FREE";   break;
        //        case request_gc:     state = "GC";     break;
        //    }
        //    printf("thid %d, current state %s\n", thid, state);
        //}

        if (request_signal[thid] == request_done){
            request_processed(type, lock, request_id, exit_signal, d_memory, 
                        dest, request_signal, request_dest, req_id);
            break;
        }
        __threadfence();
    }
}
*/

__device__
void request(request_type type,
        volatile int* exit_signal,
        RequestType& requests,
        volatile int** dest,
        int size_to_alloc){

    int thid = blockDim.x * blockIdx.x + threadIdx.x;
    int req_id = -1;
    // wait for success
    while (!exit_signal[0]){
        if (requests.request_signal[thid] == request_empty){
            post_request(type, exit_signal, requests, dest, 
                         size_to_alloc);
            break;
        }
        __threadfence();
    }
    __threadfence();

    // wait for success
    while (!exit_signal[0]){
        if (requests.request_signal[thid] == request_done){
            request_processed(type, exit_signal, requests, dest, 
                              req_id);
            break;
        }
        __threadfence();
    }
}

//consumer
__global__
void mono_app_test(//volatile int* exit_signal,
        volatile int** d_memory, 
        //RequestType& requests, 
        //volatile int* request_signal, 
        //volatile int* request_mem_size,
        //volatile int* request_id, 
        //volatile int** request_dest, 
        volatile int* exit_counter, 
        //volatile int* lock,
        int* size_to_alloc,
        int* iter_num,
        //int MONO,
        MemoryManagerType* mm){

    int thid = blockDim.x * blockIdx.x + threadIdx.x;

    for (int i=0; i<iter_num[0]; ++i){
        __threadfence();

        volatile int* new_ptr = NULL;

        d_memory[thid] = reinterpret_cast<volatile int*>(mm->malloc(4+size_to_alloc[0])); 
        d_memory[thid][0] = 0;
        new_ptr = &d_memory[thid][1];
        new_ptr[0] = thid;

        __threadfence();

        assert(d_memory[thid]);
        //int value = d_memory[thid][0];
        //if (value != 0) printf("val = %d\n", value);
        //value = d_memory[thid][1];
        assert(new_ptr[0] == thid);

        __threadfence();

        mm->free((void*)d_memory[thid]);
        __threadfence();
        d_memory[thid] = NULL;

        __threadfence();
    }

    atomicAdd((int*)&exit_counter[0], 1);

    __threadfence();

}

/*//consumer
__global__
void app_test(volatile int* exit_signal,
        volatile int** d_memory, 
        volatile int* request_signal, 
        volatile int* request_mem_size,
        volatile int* request_id, 
        volatile int** request_dest, 
        volatile int* exit_counter, 
        volatile int* lock,
        int* size_to_alloc,
        int* iter_num,
        int MONO,
        MemoryManagerType* mm){

    int thid = blockDim.x * blockIdx.x + threadIdx.x;

    for (int i=0; i<iter_num[0]; ++i){
        __threadfence();

        volatile int* new_ptr = NULL;

        request((request_type)MALLOC, exit_signal, d_memory, &new_ptr, 
                request_signal, request_mem_size, request_id, request_dest,
                lock, size_to_alloc[0]);
        new_ptr[0] = thid;

        __threadfence();

        assert(d_memory[request_id[thid]]);
        //int value = d_memory[request_id[thid]][0];
        //if (value != 0) printf("val = %d\n", value);
        assert(new_ptr[0] == thid);

        __threadfence();

        request((request_type)FREE, exit_signal, d_memory, &new_ptr,
                request_signal, request_mem_size, request_id, request_dest,
                lock, size_to_alloc[0]);

        __threadfence();
    }
    atomicAdd((int*)&exit_counter[0], 1);
    __threadfence();

}
*/

//consumer
__global__
void app_test(volatile int* exit_signal,
        volatile int* exit_counter, 
        RequestType requests,
        int* size_to_alloc,
        int* iter_num,
        int MONO,
        MemoryManagerType* mm){

    int thid = blockDim.x * blockIdx.x + threadIdx.x;
    for (int i=0; i<iter_num[0]; ++i){
        __threadfence();

        volatile int* new_ptr = NULL;

        request((request_type)MALLOC, exit_signal, requests, &new_ptr, 
                 size_to_alloc[0]);
        new_ptr[0] = thid;

        __threadfence();

        assert(requests.d_memory[requests.request_id[thid]]);
        //int value = d_memory[request_id[thid]][0];
        //if (value != 0) printf("val = %d\n", value);
        assert(new_ptr[0] == thid);

        __threadfence();

        request((request_type)FREE, exit_signal, requests, &new_ptr,
                 size_to_alloc[0]);

        __threadfence();
    }
    atomicAdd((int*)&exit_counter[0], 1);
    __threadfence();

}
/*
//consumer2
__global__
void free_app_test(volatile int* exit_signal, 
              volatile int** d_memory, 
              volatile int* request_signal, 
              volatile int* request_mem_size,
              volatile int* request_id, 
              volatile int** request_dest, 
              volatile int* exit_counter, 
              volatile int* lock,
              int size_to_alloc,
              int iter_num){
    
    __threadfence();
   
    request((request_type)FREE, exit_signal, d_memory, NULL, 
            request_signal, request_mem_size, request_id, request_dest,
            lock, 0);

    atomicAdd((int*)&exit_counter[0], 1);
}
*/
void check_persistent_kernel_results(int* exit_signal, 
                   int* exit_counter, 
                   int block_size, 
                   int app_grid_size, 
                   RequestType& requests, 
                   int requests_num,
                   bool& finish){

    // Check results
    int old_counter = -1;
    long long int iter = 0;
    long long int time_limit = 1000000000;
    //printf("waiting till allocations are done\n");
    while (iter < time_limit){
        std::this_thread::sleep_for(std::chrono::microseconds(1));
        //if (iter%60 == 0)
        //    printf("%lld min, exit counter %d\n", iter/60, exit_counter[0]);
        // Check if all allocations are done
        if (exit_counter[0] == block_size*app_grid_size){
            GUARD_CU(hipStreamSynchronize(0));
            GUARD_CU(hipPeekAtLastError());
            finish = true;
            break;
        }else{
            GUARD_CU(hipPeekAtLastError());
            if (exit_counter[0] != old_counter){
                old_counter = exit_counter[0];
                //printf("%d\n", old_counter);
                iter = 0;
            }
            ++iter;
        }
        if (iter >= time_limit){
            // Start mm and app again
            printf("time limit exceed, break\n");
            fflush(stdout);
            *exit_signal = 1;
            GUARD_CU(hipDeviceSynchronize());
            GUARD_CU(hipPeekAtLastError());
        }
    }
    GUARD_CU(hipPeekAtLastError());
}

void createStreams(hipStream_t& gc_stream, 
                   hipStream_t& mm_stream, 
                   hipStream_t& app_stream){
    GUARD_CU(hipStreamCreateWithFlags( &gc_stream, hipStreamNonBlocking));
    GUARD_CU(hipPeekAtLastError());
    GUARD_CU(hipStreamCreateWithFlags( &mm_stream, hipStreamNonBlocking));
    GUARD_CU(hipPeekAtLastError());
    GUARD_CU(hipStreamCreateWithFlags(&app_stream, hipStreamNonBlocking));
    GUARD_CU(hipPeekAtLastError());
}

void allocManaged(volatile int** ptr, size_t size){
    GUARD_CU(hipMallocManaged(ptr, size));
    GUARD_CU(hipPeekAtLastError());
    GUARD_CU(hipDeviceSynchronize());
}

void allocManaged_(int** ptr, size_t size){
    GUARD_CU(hipMallocManaged(ptr, size));
    GUARD_CU(hipPeekAtLastError());
    GUARD_CU(hipDeviceSynchronize());
}

void start_memory_manager(PerfMeasure& timing_mm, 
                          uint32_t mm_grid_size,
                          uint32_t block_size, 
                          hipCtx_t& mm_ctx,
                          volatile int* exit_signal,
                          volatile int* mm_started,
                          RequestType& requests,
                          MemoryManagerType& memory_manager){
    timing_mm.startMeasurement();
  
    auto dev_mm = memory_manager.getDeviceMemoryManager();
    void *args[] = {&exit_signal, &mm_started, &requests, &dev_mm};
    
    //GUARD_CU(hipLaunchCooperativeKernel((void*)mem_manager, mm_grid_size, block_size, args));

    GUARD_CU(hipLaunchKernel((void*)mem_manager, mm_grid_size, block_size, args));
    GUARD_CU((hipError_t)hipGetLastError());
    GUARD_CU(hipPeekAtLastError());

    timing_mm.stopMeasurement();
}

void start_garbage_collector(PerfMeasure& timing_gc, 
                          uint32_t gc_grid_size,
                          uint32_t block_size, 
                          hipCtx_t& gc_ctx,
                          volatile int* exit_signal,
                          volatile int* gc_started,
                          RequestType& requests,
                          MemoryManagerType& memory_manager){
    timing_gc.startMeasurement();
    
    auto dev_mm = memory_manager.getDeviceMemoryManager();
    void *args[] = {&exit_signal, &gc_started, &requests, &dev_mm};
    
    //GUARD_CU(hipLaunchCooperativeKernel((void*)garbage_collector, gc_grid_size, block_size, args));

    GUARD_CU(hipLaunchKernel((void*)garbage_collector, gc_grid_size, block_size, args));
    GUARD_CU((hipError_t)hipGetLastError());
    GUARD_CU(hipPeekAtLastError());
    
    timing_gc.stopMeasurement();
}

void clean_memory(uint32_t grid_size,
                  uint32_t block_size, 
                  RequestType& requests,
                  MemoryManagerType& memory_manager,
                  volatile int* exit_signal){

    *exit_signal = 1;
    GUARD_CU(hipDeviceSynchronize());
    GUARD_CU(hipPeekAtLastError());
    mem_free<<<grid_size, block_size>>>(requests.d_memory, 
            requests.request_id, 
#ifdef OUROBOROS__
            memory_manager.getDeviceMemoryManager(),
#else
#ifdef HALLOC__
            memory_manager,
#endif
#endif
            requests.requests_number);

    GUARD_CU(hipPeekAtLastError());
    GUARD_CU(hipDeviceSynchronize());
    requests.free();
    GUARD_CU(hipDeviceSynchronize());
    GUARD_CU(hipPeekAtLastError());
}


void start_application(int type, 
                       PerfMeasure& timing_sync, 
                       uint32_t grid_size,
                       uint32_t block_size, 
                       hipCtx_t& ctx,
                       volatile int* exit_signal,
                       RequestType& requests,
                       volatile int* exit_counter,
                       int* dev_size_to_alloc, 
                       int* dev_iter_num,
                       int mono, 
                       bool& kernel_complete,
                       MemoryManagerType& memory_manager){
    assert(requests.d_memory);
    fflush(stdout);
    if (mono == MPS_mono){
        printf("start application: MPS mono!\n");
        /*int* dev_size_to_alloc, *dev_iter_num;
        GUARD_CU(hipMalloc((void**)&dev_size_to_alloc, sizeof(int)));
        GUARD_CU(hipMalloc((void**)&dev_iter_num, sizeof(int)));
        GUARD_CU(hipMemset(dev_size_to_alloc, size_to_alloc, sizeof(int)));
        GUARD_CU(hipMemset(dev_iter_num, iter_num, sizeof(int)));*/
        auto dev_mm = memory_manager.getDeviceMemoryManager();
        void* args[] = {&requests.d_memory, &exit_counter, &dev_size_to_alloc, &dev_iter_num, &dev_mm};

        GUARD_CU(hipProfilerStart());
        timing_sync.startMeasurement();
        GUARD_CU(hipLaunchKernel((void*)mono_app_test, dim3(grid_size, 1, 1), dim3(block_size, 1, 1), args, 0, 0));
        /*
        mono_app_test<<<grid_size, block_size>>>(
            requests.d_memory, exit_counter, size_to_alloc, iter_num, 
            memory_manager.getDeviceMemoryManager());
        */
        GUARD_CU((hipError_t)hipCtxSynchronize());
        GUARD_CU(hipPeekAtLastError());
        timing_sync.stopMeasurement();
        GUARD_CU(hipProfilerStop());
    }else{
        debug("start applications: type %d\n", type);
        auto dev_mm = memory_manager.getDeviceMemoryManager();

        void* args[] = {&exit_signal, &exit_counter, &requests, 
                        &dev_size_to_alloc, &dev_iter_num, &mono, 
                        &dev_mm};
       /*void* args[] = {&exit_signal, &requests.d_memory, 
                        &requests.request_signal,
                        &requests.request_mem_size, 
                        &requests.request_id, 
                        &requests.request_dest, &exit_counter, 
                        &requests.lock, &dev_size_to_alloc, 
                        &dev_iter_num, &mono, &dev_mm};*/

        //GUARD_CU(hipProfilerStart());
        timing_sync.startMeasurement();
        GUARD_CU(hipLaunchKernel((void*)app_test, grid_size, block_size, args, 0, 0));
        //GUARD_CU(hipLaunchCooperativeKernel((void*)app_test, grid_size, block_size, kernelArgs));
        GUARD_CU((hipError_t)hipCtxSynchronize());
        GUARD_CU(hipPeekAtLastError());
        timing_sync.stopMeasurement();
        //GUARD_CU(hipProfilerStop());
    }

    kernel_complete = true;
}

void sync_streams(hipStream_t& gc_stream, 
                  hipStream_t& mm_stream, 
                  hipStream_t& app_stream){

    debug("waiting for streams\n");
    GUARD_CU(hipStreamSynchronize(app_stream));
    GUARD_CU(hipPeekAtLastError());
    debug("app stream synced\n");
    GUARD_CU(hipStreamSynchronize(mm_stream));
    GUARD_CU(hipPeekAtLastError());
    debug("mm stream synced\n");
    GUARD_CU(hipStreamSynchronize(gc_stream));
    GUARD_CU(hipPeekAtLastError());
    debug("gc stream synced\n");
    GUARD_CU(hipPeekAtLastError());

}


void simple_monolithic_app(int mono, int kernel_iteration_num, int size_to_alloc, size_t* ins_size, 
              size_t num_iterations, int SMs, int* sm_app, int* sm_mm, int* sm_gc, 
              int* allocs, float* uni_req_per_sec, int* array_size){

    auto instant_size = *ins_size;

    hipCtx_t default_ctx;
    GUARD_CU((hipError_t)hipCtxGetCurrent(&default_ctx));

#ifdef OUROBOROS__
    //Ouroboros initialization
    MemoryManagerType memory_manager;
    memory_manager.initialize(instant_size);
#else
#ifdef HALLOC__
    //Halloc initialization
    MemoryManagerType memory_manager(instant_size);
#endif
#endif

    GUARD_CU(hipDeviceSynchronize());
    GUARD_CU(hipPeekAtLastError());
    //Creat two asynchronous streams which may run concurrently with the default stream 0.
    //The streams are not synchronized with the default stream.
    
    volatile int* exit_signal;
    allocManaged(&exit_signal, sizeof(int32_t));

    volatile int* exit_counter;
    allocManaged(&exit_counter, sizeof(uint32_t));

    int* dev_size_to_alloc;
    int* dev_kernel_iteration_num;
    GUARD_CU(hipMallocManaged(&dev_size_to_alloc, sizeof(int)));
    GUARD_CU(hipMallocManaged(&dev_kernel_iteration_num, sizeof(int)));
    GUARD_CU(hipPeekAtLastError());
    GUARD_CU(hipDeviceSynchronize());
   
    *dev_size_to_alloc = size_to_alloc;
    *dev_kernel_iteration_num = kernel_iteration_num;
    
    hipDevice_t device;
    GUARD_CU((hipError_t)hipDeviceGet(&device, 0));
    GUARD_CU((hipError_t)hipMemPrefetchAsync((int*)dev_size_to_alloc, sizeof(int), device, NULL));
    GUARD_CU((hipError_t)hipMemPrefetchAsync((int*)dev_kernel_iteration_num, sizeof(int), device, NULL));

    int block_size = 1024;
    int app_grid_size = SMs;
    int requests_num = app_grid_size * block_size;
    
    sm_app[0] = app_grid_size;
    sm_mm[0] = 0;
    sm_gc[0] = 0;
    allocs[0] = requests_num;

    hipStream_t app_stream;
    GUARD_CU(hipStreamCreateWithFlags(&app_stream, hipStreamNonBlocking));
    GUARD_CU(hipPeekAtLastError());

    PerfMeasure /*timing_malloc_app, */malloc_total_sync;
    for (int iteration = 0; iteration < num_iterations; ++iteration){
        *exit_counter = 0;
        volatile int** d_memory{nullptr};
        GUARD_CU(hipMalloc(&d_memory, requests_num * sizeof(volatile int*)));
        GUARD_CU(hipPeekAtLastError());
        malloc_total_sync.startMeasurement();
        mono_app_test<<<app_grid_size, block_size, 0, app_stream>>>(d_memory, 
                                exit_counter, dev_size_to_alloc, dev_kernel_iteration_num, 
                                memory_manager.getDeviceMemoryManager());
        GUARD_CU(hipDeviceSynchronize());
        GUARD_CU((hipError_t)hipCtxSynchronize());
        GUARD_CU(hipPeekAtLastError());
        malloc_total_sync.stopMeasurement();
        GUARD_CU(hipFree(d_memory));
        GUARD_CU(hipDeviceSynchronize());
        GUARD_CU(hipPeekAtLastError());
    }

    auto malloc_total_sync_res = malloc_total_sync.generateResult();
    auto total_iters = kernel_iteration_num*num_iterations;
    //uni_req_per_sec[0]   = (requests_num * 1000.0)/(malloc_total_sync_res.mean_/total_iters);
    uni_req_per_sec[0] = (requests_num * 2000.0)/malloc_total_sync_res.mean_;

    printf("#measurements %d, mean %.2lf, #total iters %lu\n", malloc_total_sync_res.num_, malloc_total_sync_res.mean_, total_iters);

    printf("  %d\t\t %d\t\t %d\t\t %d\t\t %.2lf\t\t \n", requests_num, 
            app_grid_size, 0, 0, uni_req_per_sec[0]);

    *array_size = 1;

}

void mps_monolithic_app(int mono, int kernel_iteration_num, int size_to_alloc, 
            size_t* ins_size, size_t num_iterations, int SMs, int* sm_app, 
            int* sm_mm, int* sm_gc, int* allocs, float* uni_req_per_sec, 
            int* array_size){

    auto instant_size = *ins_size;

    hipCtx_t default_ctx;
    GUARD_CU((hipError_t)hipCtxGetCurrent(&default_ctx));

#ifdef OUROBOROS__
    //Ouroboros initialization
    MemoryManagerType memory_manager;
    memory_manager.initialize(instant_size);
#else
#ifdef HALLOC__
    //Halloc initialization
    MemoryManagerType memory_manager(instant_size);
#endif
#endif

    GUARD_CU(hipDeviceSynchronize());
    GUARD_CU(hipPeekAtLastError());
    
    volatile int* exit_signal;
    allocManaged(&exit_signal, sizeof(int32_t));

    volatile int* exit_counter;
    allocManaged(&exit_counter, sizeof(uint32_t));

    hipCtx_t app_ctx;
    hipDevice_t device;
    GUARD_CU((hipError_t)hipDeviceGet(&device, 0));
    GUARD_CU((hipError_t)hipCtxCreate(&app_ctx, 0, device));

    int* dev_size_to_alloc;
    int* dev_kernel_iteration_num;
    GUARD_CU(hipMallocManaged(&dev_size_to_alloc, sizeof(int)));
    GUARD_CU(hipMallocManaged(&dev_kernel_iteration_num, sizeof(int)));
    GUARD_CU(hipPeekAtLastError());
    GUARD_CU(hipDeviceSynchronize());
   
    *dev_size_to_alloc = size_to_alloc;
    *dev_kernel_iteration_num = kernel_iteration_num;
    
    GUARD_CU((hipError_t)hipMemPrefetchAsync((int*)dev_size_to_alloc, sizeof(int), device, NULL));
    GUARD_CU((hipError_t)hipMemPrefetchAsync((int*)dev_kernel_iteration_num, sizeof(int), device, NULL));

    int app_grid_size = SMs;
    int block_size = 1024;
    int requests_num = app_grid_size * block_size;
    sm_app[0] = app_grid_size;
    sm_mm[0] = 0;
    sm_gc[0] = 0;
    allocs[0] = requests_num;

    PerfMeasure /*timing_malloc_app, */malloc_total_sync;

    for (int iteration = 0; iteration < num_iterations; ++iteration){
        //printf("iter %d, requests_num %d\n", iteration, requests_num);

        *exit_signal = 0;
        *exit_counter = 0;

        GUARD_CU((hipError_t)hipMemPrefetchAsync((int*)exit_signal, sizeof(int), device, NULL));
        GUARD_CU((hipError_t)hipMemPrefetchAsync((int*)exit_counter, sizeof(int), device, NULL));

        GUARD_CU(hipDeviceSynchronize());
        GUARD_CU(hipPeekAtLastError());

        RequestType requests;
        requests.init(requests_num);
        requests.memset();

        // Run APP (all threads do malloc)
        bool kernel_complete = false;
        std::thread app_thread{[&] {
            GUARD_CU((hipError_t)hipCtxSetCurrent(app_ctx));
            //GUARD_CU((hipError_t)hipCtxSynchronize());
            debug("start app\n");
            //malloc_total_sync.startMeasurement();
            start_application(MALLOC, malloc_total_sync, 
                    app_grid_size, block_size, app_ctx, exit_signal,
                    requests, exit_counter, dev_size_to_alloc, 
                    dev_kernel_iteration_num, mono, kernel_complete, memory_manager);
            debug("app done, sync\n");
            GUARD_CU((hipError_t)hipCtxSynchronize());
            //malloc_total_sync.stopMeasurement();
            GUARD_CU(hipPeekAtLastError());
            debug("done\n");
        }};

        debug("join app\n");
        app_thread.join();
        debug("app joined\n");

        if (not kernel_complete){
            printf("kernel is not completed, free memory which app allocated\n");
            clean_memory(app_grid_size, block_size, requests, memory_manager, exit_signal);
            continue;
        }

        *exit_signal = 1;

        clean_memory(app_grid_size, block_size, requests, memory_manager, exit_signal);
        GUARD_CU(hipDeviceSynchronize());
        GUARD_CU(hipPeekAtLastError());
    }

    GUARD_CU((hipError_t)hipCtxDestroy(app_ctx));
    GUARD_CU((hipError_t)hipCtxSetCurrent(default_ctx));
    GUARD_CU(hipDeviceSynchronize());
    GUARD_CU(hipPeekAtLastError());

    auto malloc_total_sync_res = malloc_total_sync.generateResult();
    auto total_iters = kernel_iteration_num*num_iterations;
    //uni_req_per_sec[0] = (requests_num * 1000.0)/(malloc_total_sync_res.mean_/total_iters);
    uni_req_per_sec[0] = (requests_num * 2000.0)/malloc_total_sync_res.mean_;

    printf("#measurements %d, mean %.2lf, #total iters %lu\n", malloc_total_sync_res.num_, malloc_total_sync_res.mean_, total_iters);

    printf("  %d\t\t %d\t\t %d\t\t %d\t\t %.2lf\t\t \n", requests_num, 
            app_grid_size, 0, 0, uni_req_per_sec[0]);

    *array_size = 1;

}

void mps_app(int mono, int kernel_iteration_num, int size_to_alloc, 
        size_t* ins_size, size_t num_iterations, int SMs, int* sm_app, 
        int* sm_mm, int* sm_gc, int* allocs, float* uni_req_per_sec, 
        int* array_size){

    auto instant_size = *ins_size;
    hipCtx_t default_ctx;
    GUARD_CU((hipError_t)hipCtxGetCurrent(&default_ctx));

#ifdef OUROBOROS__
    //Ouroboros initialization
    MemoryManagerType memory_manager;
    memory_manager.initialize(instant_size);
#else
#ifdef HALLOC__
    //Halloc initialization
    MemoryManagerType memory_manager(instant_size);
#endif
#endif

    hipDevice_t device;
    GUARD_CU((hipError_t)hipDeviceGet(&device, 0));
    GUARD_CU(hipDeviceSynchronize());
    GUARD_CU(hipPeekAtLastError());
    //Creat two asynchronous streams which may run concurrently with the default stream 0.
    //The streams are not synchronized with the default stream.
    hipStream_t gc_stream, mm_stream, app_stream;
    createStreams(gc_stream, mm_stream, app_stream);

    volatile int* exit_signal;  allocManaged(&exit_signal, sizeof(int32_t));
    volatile int* exit_counter; allocManaged(&exit_counter, sizeof(uint32_t));
    volatile int* gc_started;   allocManaged(&gc_started, sizeof(uint32_t));
    volatile int* mm_started;   allocManaged(&mm_started, sizeof(uint32_t));

    int* dev_size_to_alloc;        allocManaged_(&dev_size_to_alloc, sizeof(int));
    int* dev_kernel_iteration_num; allocManaged_(&dev_kernel_iteration_num, 
                                                    sizeof(int));
    //GUARD_CU(hipMallocManaged(&dev_size_to_alloc, sizeof(int)));
    //GUARD_CU(hipMallocManaged(&dev_kernel_iteration_num, sizeof(int)));
    GUARD_CU(hipPeekAtLastError());
    GUARD_CU(hipDeviceSynchronize());
   
    *dev_size_to_alloc = size_to_alloc;
    *dev_kernel_iteration_num = kernel_iteration_num;
    
    GUARD_CU((hipError_t)hipMemPrefetchAsync((int*)dev_size_to_alloc, 
                                                sizeof(int), device, NULL));
    GUARD_CU((hipError_t)hipMemPrefetchAsync((int*)dev_kernel_iteration_num, 
                                                sizeof(int), device, NULL));

    int it = 0;

    int block_size = 1024;

    //SMs -= 10;

    for (int app_grid_size=1; app_grid_size<SMs; ++app_grid_size){
        //for (int app_grid_size=27; app_grid_size<28; ++app_grid_size){
    //for (int app_grid_size=1; app_grid_size<2; ++app_grid_size){
        for (int mm_grid_size=1; mm_grid_size<(SMs-app_grid_size); ++mm_grid_size){
            //for (int mm_grid_size=8; mm_grid_size<9; ++mm_grid_size){
        //for (int mm_grid_size=1; mm_grid_size<2; ++mm_grid_size){

            int gc_grid_size = SMs - app_grid_size - mm_grid_size;
            //int gc_grid_size = 1;
            if (gc_grid_size <= 0) continue;

            int requests_num{app_grid_size * block_size};

            debug("SMs: app %d, mm %d, gc %d, total %d\n", app_grid_size, mm_grid_size, gc_grid_size, SMs);
            debug("requests_num %d\n", requests_num);
            fflush(stdout);

            //output
            sm_app[it] = app_grid_size;
            sm_mm [it] = mm_grid_size;
            sm_gc [it] = gc_grid_size;
            allocs[it] = requests_num;
    
            //int mul = 1;
            int app_numBlocksPerSm = 1;// 0;
            int gc_numBlocksPerSm =  1;//0;
            int mm_numBlocksPerSm =  1;//0;

            GUARD_CU(hipOccupancyMaxActiveBlocksPerMultiprocessor(&app_numBlocksPerSm, app_test, block_size, 0));
            GUARD_CU(hipOccupancyMaxActiveBlocksPerMultiprocessor(&gc_numBlocksPerSm, mem_manager, block_size, 0));
            GUARD_CU(hipOccupancyMaxActiveBlocksPerMultiprocessor(&mm_numBlocksPerSm, garbage_collector, block_size,
            0));

            printf("num blocks per sm by cudaOccMaxActBlPerSM: app %d, mm %d, gc %d\n", 
            app_numBlocksPerSm, gc_numBlocksPerSm, mm_numBlocksPerSm);
            fflush(stdout);

            CUexecAffinityParam_v1 app_param{CUexecAffinityType::CU_EXEC_AFFINITY_TYPE_SM_COUNT, (unsigned int)app_grid_size};
            CUexecAffinityParam_v1 mm_param{CUexecAffinityType::CU_EXEC_AFFINITY_TYPE_SM_COUNT, (unsigned int)mm_grid_size};
            CUexecAffinityParam_v1 gc_param{CUexecAffinityType::CU_EXEC_AFFINITY_TYPE_SM_COUNT, (unsigned int)gc_grid_size};

            GUARD_CU(hipDeviceSynchronize());
            GUARD_CU(hipPeekAtLastError());
            GUARD_CU((hipError_t)hipGetLastError());

            auto affinity_flags = CUctx_flags::hipDeviceScheduleAuto;
            //auto affinity_flags = CUctx_flags::hipDeviceScheduleSpin;
            //auto affinity_flags = CUctx_flags::hipDeviceScheduleYield;
            //auto affinity_flags = CUctx_flags::hipDeviceScheduleBlockingSync;
            //auto affinity_flags = CUctx_flags::hipDeviceScheduleBlockingSync;
            //auto affinity_flags = CUctx_flags::hipDeviceMapHost;
            //auto affinity_flags = CUctx_flags::hipDeviceLmemResizeToMax;

            hipCtx_t app_ctx, mm_ctx, gc_ctx;
   
            GUARD_CU((hipError_t)cuCtxCreate_v3(&app_ctx,&app_param,1, affinity_flags, device));
            GUARD_CU((hipError_t)cuCtxCreate_v3(&mm_ctx, &mm_param, 1, affinity_flags, device));
            GUARD_CU((hipError_t)cuCtxCreate_v3(&gc_ctx, &gc_param, 1, affinity_flags, device));
            GUARD_CU(hipPeekAtLastError());
            GUARD_CU(hipDeviceSynchronize());
            GUARD_CU(hipPeekAtLastError());
            GUARD_CU((hipError_t)hipGetLastError());

            //Timing variables
            PerfMeasure malloc_total_sync, timing_mm, timing_gc;

            for (int iteration = 0; iteration < num_iterations; ++iteration){

                *exit_signal = 0;
                *exit_counter = 0;
                *mm_started = 0;
                *gc_started = 0;

                GUARD_CU((hipError_t)hipMemPrefetchAsync((int*)exit_signal, 
                                                    sizeof(int), device, NULL));
                GUARD_CU((hipError_t)hipMemPrefetchAsync((int*)exit_counter, 
                                                    sizeof(int), device, NULL));
                GUARD_CU((hipError_t)hipMemPrefetchAsync((int*)mm_started, 
                                                    sizeof(int), device, NULL));
                GUARD_CU((hipError_t)hipMemPrefetchAsync((int*)gc_started, 
                                                    sizeof(int), device, NULL));

                GUARD_CU(hipDeviceSynchronize());
                GUARD_CU(hipPeekAtLastError());
                GUARD_CU((hipError_t)hipGetLastError());

                RequestType requests;
                requests.init(requests_num);
                requests.memset();

                GUARD_CU(hipDeviceSynchronize());
                GUARD_CU(hipPeekAtLastError());
                GUARD_CU((hipError_t)hipGetLastError());
                
                debug("start threads\n");

                // Run Memory Manager (Presistent kernel)
                std::thread mm_thread{[&] {
                    GUARD_CU((hipError_t)hipCtxSetCurrent(mm_ctx));
                    GUARD_CU((hipError_t)hipCtxSynchronize());
                    GUARD_CU(hipPeekAtLastError());
                    debug("start mm\n");
                    start_memory_manager(timing_mm, 
                            mm_numBlocksPerSm*mm_grid_size, 
                            block_size, mm_ctx, exit_signal, 
                            mm_started, requests, memory_manager);
                    debug("mm done, sync\n");
                    GUARD_CU((hipError_t)hipCtxSynchronize());
                    GUARD_CU(hipPeekAtLastError());
                    debug("done\n");
                }};

                //std::this_thread::sleep_for(std::chrono::seconds(1));

                // Run Garbage Collector (persistent kernel)
                std::thread gc_thread{[&] {
                    GUARD_CU((hipError_t)hipCtxSetCurrent(gc_ctx));
                    GUARD_CU((hipError_t)hipCtxSynchronize());
                    GUARD_CU(hipPeekAtLastError());
                    debug("start gc\n");
                    start_garbage_collector(timing_gc, 
                            gc_numBlocksPerSm*gc_grid_size, 
                            block_size, gc_ctx, exit_signal, 
                            gc_started, requests, memory_manager);
                    debug("gc done, sync\n");
                    GUARD_CU((hipError_t)hipCtxSynchronize());
                    GUARD_CU(hipPeekAtLastError());
                    debug("done\n");
                }}; 

                //std::this_thread::sleep_for(std::chrono::seconds(1));

                //printf("-");
                //fflush(stdout);
                while (!(*gc_started && *mm_started));
                GUARD_CU((hipError_t)hipGetLastError());

                if (! (*gc_started)){
                    printf("gc did not start\n");
                }else{
                    printf("gc has started\n");
                }
                if (! (*mm_started)){
                    printf("mm did not start\n");
                }else{
                    printf("mm has started\n");
                }

                fflush(stdout);

                printf("app_numBlocksPerSm %d, app_grid_size %d, block_size %d\n", app_numBlocksPerSm, app_grid_size,
                block_size);

                // Run APP (all threads do malloc)
                bool kernel_complete = false;
                std::thread app_thread{[&] {
                    GUARD_CU((hipError_t)hipCtxSetCurrent(app_ctx));
                    GUARD_CU((hipError_t)hipCtxSynchronize());
                    GUARD_CU(hipPeekAtLastError());
                    GUARD_CU((hipError_t)hipGetLastError());
                    debug("start app\n");
                    start_application(MALLOC, malloc_total_sync, 
                            app_numBlocksPerSm*app_grid_size, 
                            block_size, app_ctx, exit_signal, 
                            requests, exit_counter, dev_size_to_alloc, 
                            dev_kernel_iteration_num, mono, 
                            kernel_complete, memory_manager);
                    GUARD_CU((hipError_t)hipCtxSynchronize());
                    GUARD_CU(hipPeekAtLastError());
                    debug("done\n");
                }};

                //std::this_thread::sleep_for(std::chrono::seconds(1));

                debug("join app\n");
                app_thread.join();
                debug("app joined\n");

                if (not kernel_complete){
                    printf("kernel is not completed, free memory which app allocated\n");
                    clean_memory(app_grid_size, block_size, requests, 
                                 memory_manager, exit_signal);
                    continue;
                }

                *exit_signal = 1;

                debug("join mm\n");
                mm_thread.join();
                debug("mm joined\n");

                debug("join gc\n");
                gc_thread.join();
                debug("gc joined\n");

                //Deallocate device memory
                //hipCtxSetCurrent(default_ctx);
                //GUARD_CU((hipError_t)hipCtxSetCurrent(default_ctx));
                clean_memory(app_grid_size, block_size, requests, memory_manager, exit_signal);
                GUARD_CU(hipDeviceSynchronize());
                GUARD_CU(hipPeekAtLastError());
            }
            printf("\n");
            debug("done\n");

            GUARD_CU((hipError_t)hipCtxDestroy(app_ctx));
            GUARD_CU((hipError_t)hipCtxDestroy(gc_ctx));
            GUARD_CU((hipError_t)hipCtxDestroy(mm_ctx));
            GUARD_CU((hipError_t)hipCtxSetCurrent(default_ctx));
            GUARD_CU(hipDeviceSynchronize());
            GUARD_CU(hipPeekAtLastError());

            // Output: the number of requests done per a second
            auto malloc_total_sync_res = malloc_total_sync.generateResult();
            auto total_iters = kernel_iteration_num*num_iterations;
            //uni_req_per_sec[it] = (requests_num * 1000.0)/(malloc_total_sync_res.mean_/total_iters);
            uni_req_per_sec[it] = (requests_num * 2000.0)/malloc_total_sync_res.mean_;

            printf("#measurements %d, mean %.2lf, #total iters %lu\n", malloc_total_sync_res.num_, malloc_total_sync_res.mean_, total_iters);

            printf("  %d\t\t %d\t\t %d\t\t %d\t\t %.2lf\t\t \n", requests_num, 
                    app_grid_size, mm_grid_size, gc_grid_size, uni_req_per_sec[it]);

            ++it;
        }
    }
    *array_size = it;
}

void pmm_init(int mono, int kernel_iteration_num, int size_to_alloc, size_t* ins_size, 
        size_t num_iterations, int SMs, int* sm_app, int* sm_mm, int* sm_gc, 
        int* allocs, float* uni_req_per_sec, int* array_size){


    printf("size to alloc per thread %d, num iterations %lu, kernel iterations %d, instantsize %lu, mono %d\n", 
            size_to_alloc, num_iterations, kernel_iteration_num, *ins_size, mono);

    std::cout << "#requests\t" << "#sm app\t\t" << "#sm mm\t\t" << "#sm gc\t\t" << "#malloc and free per sec\n";

    if (mono == MPS_mono){
        printf("MPS_mono\n");

        mps_monolithic_app(mono, kernel_iteration_num, size_to_alloc, ins_size, num_iterations, 
                SMs, sm_app, sm_mm, sm_gc, allocs, uni_req_per_sec, array_size);
    }else if (mono == simple_mono){
        printf("simple mono\n");

        simple_monolithic_app(mono, kernel_iteration_num, size_to_alloc, ins_size, num_iterations, 
                SMs, sm_app, sm_mm, sm_gc, allocs, uni_req_per_sec, array_size);
    }else{
        printf("MPS services\n");

        mps_app(mono, kernel_iteration_num, size_to_alloc, ins_size, num_iterations, 
                SMs, sm_app, sm_mm, sm_gc, allocs, uni_req_per_sec, array_size);
    }
}

}
