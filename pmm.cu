#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <cassert>
#include <algorithm>
#include <thread>
#include <chrono>
#include <any>

#include "PerformanceMeasure.cuh"
#include "hip/hip_runtime.h"
#include "pmm-utils.cuh"
#include "persistent_manager.cu"

using namespace std;

extern "C" {

void createStreams(hipStream_t& mm_stream, 
                   hipStream_t& app_stream){
    GUARD_CU(hipStreamCreateWithFlags( &mm_stream, hipStreamNonBlocking));
    GUARD_CU(hipPeekAtLastError());
    GUARD_CU(hipStreamCreateWithFlags(&app_stream, hipStreamNonBlocking));
    GUARD_CU(hipPeekAtLastError());
}

void allocManaged(volatile int** ptr, size_t size){
    GUARD_CU(hipMallocManaged(ptr, size));
    GUARD_CU(hipPeekAtLastError());
    GUARD_CU(hipDeviceSynchronize());
}

void start_memory_manager(uint32_t mm_grid_size,
                          uint32_t block_size, 
                          hipCtx_t& mm_ctx,
                          volatile int* exit_signal,
                          volatile int* mm_started,
                          RequestType& requests){
    mem_manager<<<mm_grid_size, block_size>>>(exit_signal, mm_started,
            requests.requests_number, requests.request_counter, 
            requests.request_signal, requests.request_id, requests.lock);
    GUARD_CU(hipPeekAtLastError());
}

//consumer
__global__
void mono_mock_app(request_type type, 
              volatile int* exit_signal,
              volatile int* request_signal, 
              volatile int* request_id, 
              volatile int* exit_counter, 
              volatile int* lock,
              int iter_num
              ){

    for (int i=0; i<iter_num; ++i){
        request_id[thid()] = thid();
    }
    atomicAdd((int*)&exit_counter[0], 1);
    __threadfence();
}

//consumer
__global__
void mock_app(request_type type, 
              volatile int* exit_signal,
              volatile int* request_signal, 
              volatile int* request_id, 
              volatile int* exit_counter, 
              volatile int* lock,
              int iter_num
              ){

    //int thid = blockDim.x * blockIdx.x + threadIdx.x;

    for (int i=0; i<iter_num; ++i){
        __threadfence();

        request(type, exit_signal, request_signal, lock);

        __threadfence();
    }
    
    atomicAdd((int*)&exit_counter[0], 1);

    __threadfence();
    //printf("exit counter[%d] = %d\n", thid, exit_counter[0]);
}

void start_application(request_type type, 
                       uint32_t grid_size,
                       uint32_t block_size, 
                       hipCtx_t& ctx,
                       volatile int* exit_signal,
                       RequestType& requests,
                       volatile int* exit_counter,
                       int size_to_alloc, 
                       int iter_num,
                       int mono, 
                       bool& kernel_complete){
    // Run application
    //GUARD_CU(hipPeekAtLastError());
   
    if (mono){
        mono_mock_app<<<grid_size, block_size>>>(type, exit_signal, 
                requests.request_signal, requests.request_id, exit_counter,
                requests.lock, iter_num);
    }else{
        //printf("start kernel\n");
        mock_app<<<grid_size, block_size>>>(type, exit_signal, 
                requests.request_signal, 
                requests.request_id, 
                exit_counter, 
                requests.lock,
                iter_num);
    }

    //printf("kernel done, exit counter %d\n", exit_counter[0]);
    GUARD_CU(hipPeekAtLastError());
}

void sync_streams(hipStream_t& gc_stream, 
                  hipStream_t& mm_stream, 
                  hipStream_t& app_stream){

    debug("waiting for streams\n");
    GUARD_CU(hipStreamSynchronize(app_stream));
    GUARD_CU(hipPeekAtLastError());
    debug("app stream synced\n");
    GUARD_CU(hipStreamSynchronize(mm_stream));
    GUARD_CU(hipPeekAtLastError());
    debug("mm stream synced\n");
    GUARD_CU(hipStreamSynchronize(gc_stream));
    GUARD_CU(hipPeekAtLastError());
    debug("gc stream synced\n");
    GUARD_CU(hipPeekAtLastError());

}

void mono_version(int mono, int kernel_iteration_num, int size_to_alloc, 
        size_t* ins_size, size_t num_iterations, int SMs, int* sm_app, 
        int* sm_man, int* sm_gc, int* matrix_mul, float* uni_req_per_sec, 
        int* array_size, int block_size){   
    
    hipCtx_t default_ctx;
    GUARD_CU((hipError_t)hipCtxGetCurrent(&default_ctx));
    
    volatile int* exit_signal;
    allocManaged(&exit_signal, sizeof(int32_t));

    volatile int* exit_counter;
    allocManaged(&exit_counter, sizeof(uint32_t));

    volatile int* mm_started;
    allocManaged(&mm_started, sizeof(uint32_t));

    int app_sm_size = SMs;
    int requests_num = app_sm_size * block_size;
    sm_app[0] = app_sm_size;
    sm_man[0] = 0;
    matrix_mul[0] = requests_num;

    hipCtx_t app_ctx;
    hipDevice_t device;
    GUARD_CU((hipError_t)hipDeviceGet(&device, 0));
    GUARD_CU((hipError_t)hipCtxCreate(&app_ctx, 0, device));

    GUARD_CU(hipDeviceSynchronize());
    GUARD_CU(hipPeekAtLastError());
    PerfMeasure timing_malloc_app, app_synced;

    //printf("init done\n");
    for (int iteration = 0; iteration < num_iterations; ++iteration){

        *exit_signal = 0;
        *exit_counter = 0;
        RequestType requests;
        requests.init(requests_num);
        requests.memset();

        // Run APP (all threads do malloc)
        bool kernel_complete = false;
        std::thread app_thread{[&] {
            GUARD_CU((hipError_t)hipCtxSetCurrent(app_ctx));
            debug("start app\n");
            app_synced.startMeasurement();
            start_application((request_type)MOCK, app_sm_size, block_size, 
                    app_ctx, exit_signal, requests, exit_counter, size_to_alloc, 
                    kernel_iteration_num, mono, kernel_complete);
            debug("app done, sync\n");
            GUARD_CU((hipError_t)hipCtxSynchronize());
            app_synced.stopMeasurement();
            GUARD_CU(hipPeekAtLastError());
            debug("done\n");
        }};

        //printf("join app\n");
        app_thread.join();
        //printf("app joined\n");
        *exit_signal = 1;

        //TODO: test
        GUARD_CU(hipDeviceSynchronize());
        GUARD_CU(hipPeekAtLastError());
        continue;
    }

    GUARD_CU((hipError_t)hipCtxDestroy(app_ctx));
    GUARD_CU((hipError_t)hipCtxSetCurrent(default_ctx));
    GUARD_CU(hipDeviceSynchronize());
    GUARD_CU(hipPeekAtLastError());

    auto app_synced_time = app_synced.generateResult();
    uni_req_per_sec[0] = (requests_num * 1000.0)/
        (app_synced_time.mean_/kernel_iteration_num);

    printf("  %d\t\t %d\t\t %d\t\t %.2lf\t\t \n", requests_num, 
            app_sm_size, 0, uni_req_per_sec[0]);

    *array_size = 1;
}

void pmm_init(int mono, int kernel_iteration_num, int size_to_alloc, 
        size_t* ins_size, size_t num_iterations, int SMs, int* sm_app, 
        int* sm_man, int* sm_gc, int* mock_requests, float* uni_req_per_sec, 
        int* array_size){

    printf("mono : %d\n", mono);
   
    hipCtx_t default_ctx;
    GUARD_CU((hipError_t)hipCtxGetCurrent(&default_ctx));

    //Creat one asynchronous stream which run concurrently with the default stream 0.
    //The stream is not synchronized with the default stream.
    hipStream_t mm_stream, app_stream;
    createStreams(mm_stream, app_stream);

    //int block_size = 256;
    int block_size = 1;
    int mul = 17;
    std::cout << "#requests\t" << "#sm app\t\t" << "#sm mm\t\t" 
        << "#malloc and free per sec\n";

    if (mono){
        mono_version(mono, kernel_iteration_num, size_to_alloc, ins_size, 
                num_iterations, SMs, sm_app, sm_man, sm_gc, mock_requests, 
                uni_req_per_sec, array_size, block_size);
    }else{
        printf("not mono\n");

        volatile int* exit_signal;   
        volatile int* exit_counter;  
        volatile int* mm_started;    

        allocManaged(&exit_signal, sizeof(uint32_t));
        allocManaged(&exit_counter, sizeof(uint32_t));
        allocManaged(&mm_started, sizeof(uint32_t));

        GUARD_CU(hipDeviceSynchronize());
        GUARD_CU(hipPeekAtLastError());

        int it = 0;

        for (int app_sm_size = 2; app_sm_size < 3;/*SMs;*/ ++app_sm_size){

            //int mm_sm_size = SMs - app_sm_size;
            int mm_sm_size = 2;
            int app_grid_size = mul * app_sm_size;
            int mm_grid_size = mul * mm_sm_size;
            int requests_num{app_grid_size*block_size};

            //output
            sm_app[it] = app_sm_size;
            sm_man[it] = mm_sm_size;
            mock_requests[it] = requests_num;

            printf("SM: app %d, mm %d, total %d\n", 
                                    app_sm_size, mm_sm_size, SMs);
            printf("block size %d, app grid size %d, mm grid size %d\n", 
                                    block_size, app_grid_size, mm_grid_size);
            printf("requests %d\n", requests_num);

            CUexecAffinityParam_v1 app_param{
                CUexecAffinityType::CU_EXEC_AFFINITY_TYPE_SM_COUNT, 
                (unsigned int)app_sm_size};
            CUexecAffinityParam_v1 mm_param{
                CUexecAffinityType::CU_EXEC_AFFINITY_TYPE_SM_COUNT, 
                (unsigned int)mm_sm_size};

            auto affinity_flags = CUctx_flags::hipDeviceScheduleAuto;
            hipCtx_t app_ctx, mm_ctx;
            hipDevice_t device;
            GUARD_CU((hipError_t)hipDeviceGet(&device, 0));
            GUARD_CU((hipError_t)cuCtxCreate_v3(&app_ctx, &app_param, 1, affinity_flags, device));
            GUARD_CU((hipError_t)cuCtxCreate_v3(&mm_ctx, &mm_param, 1, affinity_flags, device));
            GUARD_CU(hipDeviceSynchronize());
            GUARD_CU(hipPeekAtLastError());

            //Timing variables
            PerfMeasure app_synced;
            for (int iteration = 0; iteration < num_iterations; ++iteration){

                *exit_signal = 0;
                *exit_counter = 0;
                *mm_started = 0;

                RequestType requests;
                requests.init(requests_num);
                requests.memset();

                debug("start threads\n");

                // Run Memory Manager (Presistent kernel)
                std::thread mm_thread{[&] {
                    GUARD_CU((hipError_t)hipCtxSetCurrent(mm_ctx));
                    GUARD_CU((hipError_t)hipCtxSynchronize());
                    debug("start mm\n");
                    start_memory_manager(mm_grid_size, block_size, mm_ctx,
                            exit_signal, mm_started, requests);
                    debug("mm done, sync\n");
                    GUARD_CU((hipError_t)hipCtxSynchronize());
                    GUARD_CU(hipPeekAtLastError());
                    debug("done\n");
                }};

                printf("-");
                fflush(stdout);
                while (! *mm_started);
                printf(".");
                fflush(stdout);

                // Run APP (all threads do malloc)
                bool kernel_complete = false;
                std::thread app_thread{[&] {
                    GUARD_CU((hipError_t)hipCtxSetCurrent(app_ctx));
                    //GUARD_CU((hipError_t)hipCtxSynchronize());
                    debug("start app\n");
                    app_synced.startMeasurement();
                    start_application((request_type)MOCK, app_grid_size, block_size, 
                            app_ctx, exit_signal, requests, exit_counter, size_to_alloc, 
                            kernel_iteration_num, mono, kernel_complete);
                    debug("app done, sync\n");
                    GUARD_CU((hipError_t)hipCtxSynchronize());
                    app_synced.stopMeasurement();
                    GUARD_CU(hipPeekAtLastError());
                    debug("done\n");
                }};

                //std::this_thread::sleep_for(std::chrono::seconds(1));

                debug("join app\n");
                app_thread.join();
                debug("app joined\n");

                //*exit_signal = 1;

                debug("join mm\n");
                mm_thread.join();
                debug("mm joined\n");

                //TODO: test

                GUARD_CU(hipDeviceSynchronize());
                GUARD_CU(hipPeekAtLastError());
                GUARD_CU((hipError_t)hipCtxSetCurrent(default_ctx));
                GUARD_CU((hipError_t)hipCtxSynchronize());
            }
            printf("\n");
            debug("done\n");

            GUARD_CU((hipError_t)hipCtxDestroy(app_ctx));
            GUARD_CU((hipError_t)hipCtxDestroy(mm_ctx));
            GUARD_CU((hipError_t)hipCtxSetCurrent(default_ctx));
            GUARD_CU(hipDeviceSynchronize());
            GUARD_CU(hipPeekAtLastError());

            // Output: the number of requests done per a second
            auto app_synced_time = app_synced.generateResult();
            uni_req_per_sec[it]  = (requests_num * 1000.0)/(app_synced_time.mean_/kernel_iteration_num);

            printf("  %d\t\t %d\t\t %d\t\t %.2lf\t\t \n", requests_num, 
                    app_sm_size, mm_sm_size, uni_req_per_sec[it]);

            ++it;
        }
        *array_size = it;
        }
    }
}
